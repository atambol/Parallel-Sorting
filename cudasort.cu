
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <vector>
#include <limits>

#define THREADS 512

#ifdef __cplusplus
extern "C"
{
#endif

using namespace std; 

__global__ void insertion_sort(float *device_data, int *device_start, int *device_offset){
    int start=device_start[blockIdx.x];
    int offset=device_offset[blockIdx.x];
    device_data += start;
    float tmp;
    int i, j, k;

    // Perform insertion sort
    for (i = 0; i < offset-1; i++) {
        j = i + 1;
        k = i;

        // find the smallest element
        for (j = i+1; j < offset; j++)
            if (device_data[k] > device_data[j])
                k = j;

        // swap elements
        tmp=device_data[k];
        device_data[k]=device_data[i];
        device_data[i]=tmp;
    }
}

std::vector<vector <float> > get_buckets(int bucket_count) {
	std::vector<vector <float> > buckets;
    int i;
    for (i = 0;i < bucket_count; i++) {
        std::vector<float> list;
        buckets.push_back(list);
    }
    return buckets;
}

float get_max(int number_of_elements, float *data) {
	float max = -std::numeric_limits<float>::infinity();
    int i;
    for (i = 0; i < number_of_elements; i++)
        if(max < data[i])
            max = data[i];
    return max;
}

std::vector<vector <float> > assign_bucket(int number_of_elements, float *data, float max, int bucket_count) {
	std::vector<vector <float> > buckets = get_buckets(bucket_count);
	int index;
    int i;
	for (i = 0 ;i < number_of_elements; i++){
        index = int((bucket_count*data[i])/(max+0.01)); // same as used in pthreads
        buckets[index].push_back(data[i]);
    }
    return buckets;
}

void bucket_sort(std::vector<vector <float> > buckets, int bucket_count, 
					float *data, int *start, int *offset) {
	int size, i, j;
	int index = 0;
	for (i=0; i < bucket_count; i++) {
		size = buckets[i].size();
		offset[i]=int(buckets[i].size());
		start[i]=int(index);
		for (j=0; j < size; j++){
			data[index]=float(buckets[i][j]);
			index++;
		}
	}
}

int cuda_sort(int number_of_elements, float *a)
{
	int bucket_count;
	float *data, max;
	int *start, *offset;
	std::vector<vector <float> > buckets;

	// get bucket count - best performance found at this configuration
    bucket_count=number_of_elements/64;

	// find max element
    max = get_max(number_of_elements, a);

    // assign elements to appropriate buckets 
    buckets = assign_bucket(number_of_elements, a, max, bucket_count);

    // perform bucket sort on the array by arranging the bucket elements
	data = (float *)malloc(number_of_elements*sizeof(float));
	start = (int *)malloc(bucket_count*sizeof(int));
	offset = (int *)malloc(bucket_count*sizeof(int));
	bucket_sort(buckets, bucket_count, data, start, offset);

	// prepare for running insertion sorting on GPU in parallel
    float *device_data;
    int *device_start, *device_offset;
    hipMalloc((void **) &device_data, sizeof(float)*number_of_elements);
    hipMemcpy(device_data, data, sizeof(float)*number_of_elements, hipMemcpyHostToDevice);
    hipMalloc((void **) &device_start, sizeof(int)*bucket_count);
    hipMemcpy(device_start, start, sizeof(int)*bucket_count, hipMemcpyHostToDevice);
    hipMalloc((void **) &device_offset, sizeof(int)*bucket_count);
    hipMemcpy(device_offset, offset, sizeof(int)*bucket_count, hipMemcpyHostToDevice);

    // run sorting on GPU
    dim3 dimGrid(bucket_count);
    dim3 dimBlock(1);
    insertion_sort<<<dimGrid, dimBlock>>>(device_data, device_start, device_offset);

    // copy results
    hipMemcpy(a, device_data, sizeof(float)*number_of_elements, hipMemcpyDeviceToHost);

    // free back to heap
    hipFree(device_data);
    hipFree(device_start);
    hipFree(device_offset);
    free(data);
    free(start);
    free(offset);
	return 0;
}

#ifdef __cplusplus
}
#endif

